#include "hip/hip_runtime.h"
/// @author Quang Nguyen nguyenqu2
//#define _GNU_SOURCE

#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <pthread.h>
#include <math.h>

#include "gpu-md5.cu"

#define MAX_USERNAME_LENGTH 64
#define PASSWORD_LENGTH 6
#define PASSWORD_SPACE_SIZE 308915776
#define NUM_THREADS 512
#define CHAR_NUM 26
#define CRACKED 1
#define NOT_CRACKED -1

/******************* Device code **************************/
/**
 * This cuntion run on device to crack the code. The idea is that it generate a candidate password,
 * find its hashcode and compare it with input_hash. If we find out, we print the result to output,
 * set the cracked variable.
 * \param input_hash the given hash, belong to the password we need to crack.
 * \param output the correct password. We need to print value into this string
 * \param cracked the number to indicate whether we crack the code.
 * \param id_offset this is the number of passwords we check, act as an offset for N.
 *      this decides the candidate password.
 */
__global__ void single_crack_MD5(uint8_t *input_hash, char* output, int *cracked, int id_offset) {
    if (*cracked == NOT_CRACKED) {
        // get N based on the number id of block. This is used to construct to candidate password.
        // N = 0 would give us "aaaaaa"
        // N = 1 would give us "aaaaab" so on.
        int N = threadIdx.x + blockIdx.x * blockDim.x + id_offset;
        if (N >= PASSWORD_SPACE_SIZE) {
            return;
        } 

        // generate candidate based on N
        char *candidate_password = (char *) malloc(sizeof(char) * (PASSWORD_LENGTH + 1));
        char temp[] = "aaaaaa";
        memcpy(candidate_password, temp, sizeof(char) * (PASSWORD_LENGTH + 1));
        for (int j = PASSWORD_LENGTH - 1; j >= 0; j--) {
            candidate_password[j] = (char) ('a' + N % CHAR_NUM);
            N = N / CHAR_NUM;
        }

        // generate candidate hash
        uint8_t *candidate_hash = (uint8_t*) malloc(sizeof(uint8_t) * MD5_UNSIGNED_HASH_LEN);
        md5((unsigned char*) candidate_password, PASSWORD_LENGTH, candidate_hash);

        // compare candidate hash with input hash
        for (int i = 0; i < MD5_UNSIGNED_HASH_LEN; i++) {
            if (input_hash[i] != candidate_hash[i]) {

                free(candidate_password);
                free(candidate_hash);
                return;
            }
        }
        
        // update cracked
        *cracked = CRACKED;
        memcpy(output, candidate_password, sizeof(char) * (PASSWORD_LENGTH + 1));

        free(candidate_password);
        free(candidate_hash);
    }
}


/******************** Password crack code *****************/
/**
 * This function call the gpu function to crack code. Each time, we test 1000 * 500 passwords until
 * we check all the password space.
 * \param input_hash the given hash, belong to the password we need to crack.
 * \param output the correct password. We need to print value into this string
 * \param cracked the number to indicate whether we crack the code.
 */
void crack_single_password(uint8_t *input_hash, char *output, int *cracked) {
    int num_block = 10000;
    int block_size = 500;

    int tested_passwords = 0;

    // testing the each password
    while (tested_passwords < PASSWORD_SPACE_SIZE) {
        if (*cracked == NOT_CRACKED) {
            // call the gpu function
            single_crack_MD5<<<num_block, block_size>>>(input_hash, output, cracked, tested_passwords);
            hipDeviceSynchronize();

            tested_passwords += num_block * block_size;
        } else {
            break;
        }
    }
}


/******************** Provided Code ***********************/
/**
 * Convert a string representation of an MD5 hash to a sequence
 * of bytes. The input md5_string must be 32 characters long, and
 * the output buffer bytes must have room for MD5_DIGEST_LENGTH
 * bytes.
 *
 * \param md5_string  The md5 string representation
 * \param bytes       The destination buffer for the converted md5 hash
 * \returns           0 on success, -1 otherwise
 */
int md5_string_to_bytes(const char *md5_string, uint8_t *hash_code) {
    // Check for a valid MD5 string
    if (strlen(md5_string) != 2 * MD5_UNSIGNED_HASH_LEN) return -1;

    // Start our "cursor" at the start of the string
    const char *pos = md5_string;

    // Loop until we've read enough bytes
    for (size_t i = 0; i < MD5_UNSIGNED_HASH_LEN; i++) {
        // Read one byte (two characters)
        int rc = sscanf(pos, "%2hhx", &hash_code[i]);
        if (rc != 1) return -1;

        // Move the "cursor" to the next hexadecimal byte
        pos += 2;
    }
    return 0;
}

void print_usage(const char *exec_name) {
    fprintf(stderr, "Usage:\n");
    fprintf(stderr, "  %s single <MD5 hash>\n", exec_name);
    fprintf(stderr, "  %s list <password file name>\n", exec_name);
}

int main(int argc, char **argv) {
    // check the input arguments' correctness
    if (argc != 3) {
        print_usage(argv[0]);
        exit(1);
    }

    if (strcmp(argv[1], "single") == 0) {
        // allocate variable to use on device and host
        uint8_t *input_hash;
        hipMallocManaged(&input_hash, sizeof(uint8_t) * MD5_UNSIGNED_HASH_LEN);

        int *cracked;
        hipMallocManaged(&cracked, sizeof(int));
        *cracked = NOT_CRACKED;

        // The input MD5 hash is a string in hexadecimal. Convert it to bytes.
        if (md5_string_to_bytes(argv[2], input_hash)) {
            fprintf(stderr, "Input has value %s is not a valid MD5 hash.\n", argv[2]);

            // Early exit. Free variable
            hipFree(input_hash);
            hipFree(cracked);
            exit(1);
        }

        // Now call the crack_single_password function
        // result hold the correct password.
        char *result;
        hipMallocManaged(&result, sizeof(char) * (PASSWORD_LENGTH + 1));

        // call crack single password
        crack_single_password (input_hash, result, cracked);

        // check if we successfully cracked the password
        if (*cracked == NOT_CRACKED) {
            printf("No matching password found.\n");
        } else {
            printf("%s\n", result);
        }

        // Free variable
        hipFree(result);
        hipFree(input_hash);
        hipFree(cracked);

    } else if (strcmp(argv[1], "list") == 0) {
    //     // Make and initialize a password set
    //     password_set_t passwords;
    //     init_password_set(&passwords);

    //     // Open the password file
    //     FILE *password_file = fopen(argv[2], "r");
    //     if (password_file == NULL) {
    //         perror("opening password file");
    //         exit(2);
    //     }

    //     int password_count = 0;

    //     // Read until we hit the end of the file
    //     while (!feof(password_file)) {
    //         // Make space to hold the username
    //         char username[MAX_USERNAME_LENGTH];

    //         // Make space to hold the MD5 string
    //         char md5_string[MD5_UNSIGNED_HASH_LEN * 4 + 1];

    //         // Make space to hold the MD5 bytes
    //         unsigned password_hash[MD5_UNSIGNED_HASH_LEN];

    //         // Try to read. The space in the format string is required to eat the newline
    //         if (fscanf(password_file, "%s %s ", username, md5_string) != 2) {
    //             fprintf(stderr, "Error reading password file: malformed line\n");
    //             exit(2);
    //         }

    //         // Convert the MD5 string to MD5 bytes in our new node
    //         if (md5_string_to_unsigned(md5_string, password_hash) != 0) {
    //             fprintf(stderr, "Error reading MD5\n");
    //             exit(2);
    //         }

    //         // Add the password to the password set
    //         add_password(&passwords, username, password_hash);
    //         password_count++;
    //     }


    //     // Now run the password list cracker
    //     int cracked = crack_password_list(&passwords);

    //     printf("Cracked %d of %d passwords.\n", cracked, password_count);

    //     // free passwords set
    //     user_password_t *cursor = passwords.head;
    //     user_password_t *temp = NULL;
    //     while (cursor != NULL) {
    //         temp = cursor;
    //         cursor = cursor->next;
    //         free(temp);
    //     }

    // } else {
    //     print_usage(argv[0]);
    //     exit(1);
    }
    return 0;
}
