#include "hip/hip_runtime.h"
/// @author Quang Nguyen nguyenqu2
//#define _GNU_SOURCE

#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <pthread.h>
#include <math.h>

#include "gpu_md5.cu"

#define MAX_USERNAME_LENGTH 64
#define PASSWORD_LENGTH 6
#define PASSWORD_SPACE_SIZE 308915776
#define NUM_THREADS 512
#define CHAR_NUM 26
#define CRACKED 1
#define NOT_CRACKED -1


/************** SUPPORT STRUCTURE *************************/
/**
 * This is a node in password_set_t that contains information about a user and their password.
 */
typedef struct user_password {
    char username[MAX_USERNAME_LENGTH];
    unsigned password_hash[4];
    int cracked_password;

    struct user_password *next;
} user_password_t;

/**
 * This struct is the root of the data structure that will hold users and hashed passwords.
 * This could be any type of data structure you choose: list, array, tree, hash table, etc.
 * Implement this data structure for part B of the lab.
 */
typedef struct password_set {
    user_password_t *head;
    int user_num;
} password_set_t;

/******************* Device code **************************/

__global__ void single_crack_MD5(uint8_t *input_hash, char* output, int *cracked, int id_offset) {
    if (*cracked == NOT_CRACKED) {
        int N = threadIdx.x + blockIdx.x * blockDim.x;
        if (N >= PASSWORD_SPACE_SIZE) {
            return;
        } 

        // generate candidate based on N
        char *candidate_password = (char *) malloc(sizeof(char) * (PASSWORD_LENGTH + 1));
        char temp[] = "aaaaaa";
        memcpy(candidate_password, temp, sizeof(char) * (PASSWORD_LENGTH + 1));
        for (int j = PASSWORD_LENGTH - 1; j >= 0; j--) {
            candidate_password[j] = (char) ('a' + N % CHAR_NUM);
            N = N / CHAR_NUM;
        }

        // generate candidate hash
        uint8_t *candidate_hash = (uint8_t*) malloc(sizeof(uint8_t) * MD5_UNSIGNED_HASH_LEN);
        md5((unsigned char*) candidate_password, PASSWORD_LENGTH, candidate_hash);

        // compare candidate hash with input hash
        if (memcmp(candidate_hash, input_hash, sizeof(uint8_t) * MD5_UNSIGNED_HASH_LEN) != 0) {
            free(candidate_password);
            free(candidate_hash);
            return;
        } else {
            // update cracked
            atomicAdd(cracked, N + 1);
            memcpy(output, candidate_password, sizeof(char) * (PASSWORD_LENGTH + 1));

            free(candidate_password);
            free(candidate_hash);
        }
    }
}


/******************** Password crack code *****************/
void crack_single_password(uint8_t *input_hash, char *output, int *cracked) {
    int num_block = 1000;
    int block_size = 500;

    int total_thread = 0;

    while (total_thread < PASSWORD_SPACE_SIZE) {
        if (*cracked == NOT_CRACKED) {
            single_crack_MD5<<<num_block, block_size>>>(input_hash, output, cracked, total_thread);
            hipDeviceSynchronize();

            total_thread += num_block * block_size;
        } else {
            break;
        }
    }

    int temp = *cracked;
    if (temp != NOT_CRACKED) {
        strcpy(output, "aaaaaa");
        for (int j = PASSWORD_LENGTH - 1; j >= 0; j--) {
            output[j] = (char) ('a' + temp % CHAR_NUM);
            temp = temp / CHAR_NUM;
        }
    }
}


/******************** Provided Code ***********************/
/**
 * Convert a string representation of an MD5 hash to a sequence
 * of bytes. The input md5_string must be 32 characters long, and
 * the output buffer bytes must have room for MD5_DIGEST_LENGTH
 * bytes.
 *
 * \param md5_string  The md5 string representation
 * \param bytes       The destination buffer for the converted md5 hash
 * \returns           0 on success, -1 otherwise
 */
int md5_string_to_bytes(const char *md5_string, uint8_t *hash_code) {
    // Check for a valid MD5 string
    if (strlen(md5_string) != 2 * MD5_UNSIGNED_HASH_LEN) return -1;

    // Start our "cursor" at the start of the string
    const char *pos = md5_string;

    // Loop until we've read enough bytes
    for (size_t i = 0; i < MD5_UNSIGNED_HASH_LEN; i++) {
        // Read one byte (two characters)
        int rc = sscanf(pos, "%2hhx", &hash_code[i]);
        if (rc != 1) return -1;

        // Move the "cursor" to the next hexadecimal byte
        pos += 2;
    }
    return 0;
}

void print_usage(const char *exec_name) {
    fprintf(stderr, "Usage:\n");
    fprintf(stderr, "  %s single <MD5 hash>\n", exec_name);
    fprintf(stderr, "  %s list <password file name>\n", exec_name);
}

int main(int argc, char **argv) {
    if (argc != 3) {
        print_usage(argv[0]);
        exit(1);
    }

    if (strcmp(argv[1], "single") == 0) {
        uint8_t *input_hash;
        hipMallocManaged(&input_hash, sizeof(unsigned) * MD5_UNSIGNED_HASH_LEN);

        int *cracked;
        hipMallocManaged(&cracked, sizeof(int));
        *cracked = NOT_CRACKED;

        // The input MD5 hash is a string in hexadecimal. Convert it to bytes.
        if (md5_string_to_bytes(argv[2], input_hash)) {
            fprintf(stderr, "Input has value %s is not a valid MD5 hash.\n", argv[2]);

            // Free variable
            hipFree(input_hash);
            hipFree(cracked);
            exit(1);
        }

        // Now call the crack_single_password function
        char *result = (char *) malloc(sizeof(char) * (PASSWORD_LENGTH + 1));
        crack_single_password (input_hash, result, cracked);
        if (*cracked == NOT_CRACKED) {
            printf("No matching password found.\n");
        } else {
            printf("%s\n", result);
        }

        // Free variable
        hipFree(input_hash);
        hipFree(cracked);

    } else if (strcmp(argv[1], "list") == 0) {
    //     // Make and initialize a password set
    //     password_set_t passwords;
    //     init_password_set(&passwords);

    //     // Open the password file
    //     FILE *password_file = fopen(argv[2], "r");
    //     if (password_file == NULL) {
    //         perror("opening password file");
    //         exit(2);
    //     }

    //     int password_count = 0;

    //     // Read until we hit the end of the file
    //     while (!feof(password_file)) {
    //         // Make space to hold the username
    //         char username[MAX_USERNAME_LENGTH];

    //         // Make space to hold the MD5 string
    //         char md5_string[MD5_UNSIGNED_HASH_LEN * 4 + 1];

    //         // Make space to hold the MD5 bytes
    //         unsigned password_hash[MD5_UNSIGNED_HASH_LEN];

    //         // Try to read. The space in the format string is required to eat the newline
    //         if (fscanf(password_file, "%s %s ", username, md5_string) != 2) {
    //             fprintf(stderr, "Error reading password file: malformed line\n");
    //             exit(2);
    //         }

    //         // Convert the MD5 string to MD5 bytes in our new node
    //         if (md5_string_to_unsigned(md5_string, password_hash) != 0) {
    //             fprintf(stderr, "Error reading MD5\n");
    //             exit(2);
    //         }

    //         // Add the password to the password set
    //         add_password(&passwords, username, password_hash);
    //         password_count++;
    //     }


    //     // Now run the password list cracker
    //     int cracked = crack_password_list(&passwords);

    //     printf("Cracked %d of %d passwords.\n", cracked, password_count);

    //     // free passwords set
    //     user_password_t *cursor = passwords.head;
    //     user_password_t *temp = NULL;
    //     while (cursor != NULL) {
    //         temp = cursor;
    //         cursor = cursor->next;
    //         free(temp);
    //     }

    // } else {
    //     print_usage(argv[0]);
    //     exit(1);
    }
    return 0;
}