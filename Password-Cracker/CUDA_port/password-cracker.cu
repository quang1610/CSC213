#include "hip/hip_runtime.h"
/// @author Quang Nguyen nguyenqu2
#define _GNU_SOURCE

#include <openssl/md5.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <pthread.h>
#include <hip/hip_runtime.h>

#define MAX_USERNAME_LENGTH 64
#define PASSWORD_LENGTH 6
#define PASSWORD_SPACE_SIZE 308915776
#define NUM_THREADS 512
#define CHAR_NUM 26
#define CRACKED 1
#define NOT_CRACKED -1


/************** SUPPORT STRUCTURE *************************/
/**
 * This is a node in password_set_t that contains information about a user and their password.
 */
typedef struct user_password {
    char username[MAX_USERNAME_LENGTH];
    uint8_t password_hash[MD5_DIGEST_LENGTH];
    int cracked_password;

    struct user_password *next;
} user_password_t;

/**
 * This struct is the root of the data structure that will hold users and hashed passwords.
 * This could be any type of data structure you choose: list, array, tree, hash table, etc.
 * Implement this data structure for part B of the lab.
 */
typedef struct password_set {
    user_password_t *head;
    int user_num;
} password_set_t;

/******************* Device code **************************/



/******************** Password crack code *****************/
void crack_single_password(uint8_t *input_hash, char *output, short *cracked) {

    if *cracked != CRACKED {
        int i = threadIdx.x + blockIdx.x * NUM_THREADS;

        char candidate_password = "aaaaaa";
        for (int j = PASSWORD_LENGTH - 1; j >= 0; j--) {
            candidate_password[j] = (char) ('a' + i % CHAR_NUM);
            i = i / CHAR_NUM;
        }
    
        // checking password hash
        uint8_t candidate_hash[MD5_DIGEST_LENGTH]; //< This will hold the hash of the candidate password
        MD5((unsigned char *) candidate_password, strlen(candidate_password), candidate_hash); //< Do the hash
    
        // Now check if the hash of the candidate password matches the input hash
        if (memcmp(input_hash, candidate_hash, MD5_DIGEST_LENGTH) == 0) {
            // Match! Copy the password to the output and return 0 (success)
            memcpy(output, candidate_password, sizeof(char) * (PASSWORD_LENGTH + 1));
            atomicAdd(*cracked, 0 - *cracked + CRACKED);
        }
    }
}


/******************** Provided Code ***********************/
/**
 * Convert a string representation of an MD5 hash to a sequence
 * of bytes. The input md5_string must be 32 characters long, and
 * the output buffer bytes must have room for MD5_DIGEST_LENGTH
 * bytes.
 *
 * \param md5_string  The md5 string representation
 * \param bytes       The destination buffer for the converted md5 hash
 * \returns           0 on success, -1 otherwise
 */
int md5_string_to_bytes(const char *md5_string, uint8_t *bytes) {
    // Check for a valid MD5 string
    if (strlen(md5_string) != 2 * MD5_DIGEST_LENGTH) return -1;

    // Start our "cursor" at the start of the string
    const char *pos = md5_string;

    // Loop until we've read enough bytes
    for (size_t i = 0; i < MD5_DIGEST_LENGTH; i++) {
        // Read one byte (two characters)
        int rc = sscanf(pos, "%2hhx", &bytes[i]);
        if (rc != 1) return -1;

        // Move the "cursor" to the next hexadecimal byte
        pos += 2;
    }

    return 0;
}

void print_usage(const char *exec_name) {
    fprintf(stderr, "Usage:\n");
    fprintf(stderr, "  %s single <MD5 hash>\n", exec_name);
    fprintf(stderr, "  %s list <password file name>\n", exec_name);
}

int main(int argc, char **argv) {
    if (argc != 3) {
        print_usage(argv[0]);
        exit(1);
    }

    if (strcmp(argv[1], "single") == 0) {
        // The input MD5 hash is a string in hexadecimal. Convert it to bytes.
        uint8_t *input_hash;
        hipMallocManaged(sizeof(uint8_t) * MD5_DIGEST_LENGTH);
        if (md5_string_to_bytes(argv[2], input_hash)) {
            fprintf(stderr, "Input has value %s is not a valid MD5 hash.\n", argv[2]);
            exit(1);
        }

        // Now call the crack_single_password function
        char *result;
        short *cracked;
        hipMallocManaged(result, sizeof(char) * (PASSWORD_LENGTH + 1));
        hipMallocManaged(cracked, sizeof(short));
        *cracked = NOT_CRACKED;

        int num_block = PASSWORD_SPACE_SIZE / NUM_THREADS + 1
        int num_thread = NUM_THREADS
        crack_single_password (input_hash, result, cracked);
        if (cracked == NOT_CRACKED) {
            printf("No matching password found.\n");
        } else {
            printf("%s\n", result);
        }

        hipFree(result);
        hipFree(input_hash);
        hipFree(cracked);

    } else if (strcmp(argv[1], "list") == 0) {
        // Make and initialize a password set
        password_set_t passwords;
        init_password_set(&passwords);

        // Open the password file
        FILE *password_file = fopen(argv[2], "r");
        if (password_file == NULL) {
            perror("opening password file");
            exit(2);
        }

        int password_count = 0;

        // Read until we hit the end of the file
        while (!feof(password_file)) {
            // Make space to hold the username
            char username[MAX_USERNAME_LENGTH];

            // Make space to hold the MD5 string
            char md5_string[MD5_DIGEST_LENGTH * 2 + 1];

            // Make space to hold the MD5 bytes
            uint8_t password_hash[MD5_DIGEST_LENGTH];

            // Try to read. The space in the format string is required to eat the newline
            if (fscanf(password_file, "%s %s ", username, md5_string) != 2) {
                fprintf(stderr, "Error reading password file: malformed line\n");
                exit(2);
            }

            // Convert the MD5 string to MD5 bytes in our new node
            if (md5_string_to_bytes(md5_string, password_hash) != 0) {
                fprintf(stderr, "Error reading MD5\n");
                exit(2);
            }

            // Add the password to the password set
            add_password(&passwords, username, password_hash);
            password_count++;
        }


        // Now run the password list cracker
        int cracked = crack_password_list(&passwords);

        printf("Cracked %d of %d passwords.\n", cracked, password_count);

        // free passwords set
        user_password_t *cursor = passwords.head;
        user_password_t *temp = NULL;
        while (cursor != NULL) {
            temp = cursor;
            cursor = cursor->next;
            free(temp);
        }

    } else {
        print_usage(argv[0]);
        exit(1);
    }
    return 0;
}