#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "gpu_md5.cu"

#define PASSWORD_LENGTH 6
#define MD5_UNSIGNED_HASH_LEN 16

// This program take a string and return a hashcode for it. 

__global__ void get_md5_hashcode(unsigned char *password, int password_len, unsigned *hash_code) {
    md5((unsigned char*) password, PASSWORD_LENGTH, hash_code);
}

int main(int argc, char **argv) {
    if (argc != 2) {
        printf("The program require an argument, a string, to work!\n");
        exit(1);
    }

    if (strlen(argv[1]) != PASSWORD_LENGTH) {
        printf("The password's length must be %d\n", PASSWORD_LENGTH);
        exit(1);
    }

    unsigned char *hash_code;
    hipMallocManaged(&hash_code, sizeof(unsigned char) * (MD5_UNSIGNED_HASH_LEN + 1));

    char *gpu_password;
    hipMalloc(&gpu_password, sizeof(char) * (PASSWORD_LENGTH + 1));
    hipMemcpy(gpu_password, argv[1], sizeof(char) * (PASSWORD_LENGTH + 1), hipMemcpyHostToDevice);

    get_md5_hashcode<<<1,1>>>((unsigned char *) gpu_password, PASSWORD_LENGTH, hash_code);
    hipDeviceSynchronize();

    // print the pass code
    printf("%u\n", hash_code[i]);

    hipFree(hash_code);
    hipFree(gpu_password);

    return 0;
}